#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include<stdio.h>
#include <cmath>
#include<stdlib.h>
#include<iostream>
#include <iomanip>
#include <math.h> 
#include <stdio.h>

using namespace std;
typedef unsigned int uint;



static const int wholeArraySize = 4;
static const int blockSize = 1024;
static const int gridSize = 16;

__device__ bool lastBlock(int* counter) {
    __threadfence(); //ensure that partial result is visible by all blocks
    int last = 0;
    if (threadIdx.x == 0)
        last = atomicAdd(counter, 1);
    return __syncthreads_or(last == gridDim.x - 1);
}

__global__ void sumCommMultiBlock(
    int start, 
    int end, 
    int base, 
    int power, 
    int modulus,
    int* gOut, 
    int* lastBlockCounter){

    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;

    int sum = 0;
    for (int i = start + gthIdx; i < end; i += gridSize)

        int res = pow(base, i);

        if (fmod(res, modulus) == power)
            sum += i;


    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
    if (lastBlock(lastBlockCounter)) {
        shArr[thIdx] = thIdx < gridSize ? gOut[thIdx] : 0;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size)
                shArr[thIdx] += shArr[thIdx + size];
            __syncthreads();
        }
        if (thIdx == 0)
            gOut[0] = shArr[0];
    }
}

int sumArray(int* arr) {
    int* dev_arr;

    //cout << "start alloc" << endl;
    hipMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    hipMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), hipMemcpyHostToDevice);

    int base = 7;
    int power = 15;
    int modulus = 41;
    //hipMalloc((void**)&target, sizeof(int));
    //hipMemcpy(target, 10, sizeof(int), hipMemcpyHostToDevice);

    int out;
    int* dev_out;
    hipMalloc((void**)&dev_out, sizeof(int) * gridSize);

    int* dev_lastBlockCounter;
    hipMalloc((void**)&dev_lastBlockCounter, sizeof(int));
    hipMemset(dev_lastBlockCounter, 0, sizeof(int));



    sumCommMultiBlock << <gridSize, blockSize >> > (1, 4, 7, 15, 41, dev_out, dev_lastBlockCounter);
    hipDeviceSynchronize();

    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_arr);
    hipFree(dev_out);

    return out - gridSize;
}


int main() {

    const int arraySize = 4;
    int a[arraySize] = { 1,  2,  3,  4 };


    cout << sumArray(a) << endl;
}


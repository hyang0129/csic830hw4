#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include<stdio.h>
#include <cmath>
#include<stdlib.h>
#include<iostream>
#include <iomanip>
#include <math.h> 
#include <stdio.h>

using namespace std;
typedef unsigned int uint;



static const int wholeArraySize = 4;
static const int blockSize = 1024;
static const int gridSize = 16;

__device__ bool lastBlock(int* counter) {
    __threadfence(); //ensure that partial result is visible by all blocks
    int last = 0;
    if (threadIdx.x == 0)
        last = atomicAdd(counter, 1);
    return __syncthreads_or(last == gridDim.x - 1);
}

__global__ void sumCommMultiBlock(
    int start,
    int end,
    int base,
    int power,
    int modulus,
    int* gOut,
    int* lastBlockCounter) {

    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;

    int sum = 0;
    long long int res = 0;
    for (int i = start + gthIdx; i < end; i += gridSize)
        {
            res = pow(base, i);

            if ((res % modulus) == power) {
                sum += i;
            }

            if (i == 10) {
                sum += res;
            }
        }



    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
    if (lastBlock(lastBlockCounter)) {
        shArr[thIdx] = thIdx < gridSize ? gOut[thIdx] : 0;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size)
                shArr[thIdx] += shArr[thIdx + size];
            __syncthreads();
        }
        if (thIdx == 0)
            gOut[0] = shArr[0];
    }
}

int sumArray(int* arr) {
    int* dev_arr;

    //cout << "start alloc" << endl;
    hipMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    hipMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), hipMemcpyHostToDevice);

    int base = 7;
    int power = 15;
    int modulus = 41;

    base = 68093; 
    power = 836856; 
    modulus = 10000019;
    //hipMalloc((void**)&target, sizeof(int));
    //hipMemcpy(target, 10, sizeof(int), hipMemcpyHostToDevice);

    int out;
    int* dev_out;
    hipMalloc((void**)&dev_out, sizeof(int) * gridSize);

    int* dev_lastBlockCounter;
    hipMalloc((void**)&dev_lastBlockCounter, sizeof(int));
    hipMemset(dev_lastBlockCounter, 0, sizeof(int));



    sumCommMultiBlock << <gridSize, blockSize >> > (1, 
        20, base, power, modulus, dev_out, dev_lastBlockCounter);
    hipDeviceSynchronize();

    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_arr);
    hipFree(dev_out);

    return out - gridSize;
}

// Returns minimum x for which a ^ x % m = b % m, a and m are coprime.
int solve(int a, int b, int m) {
    a %= m, b %= m;
    int n = sqrt(m) + 1;

    int an = 1;
    for (int i = 0; i < n; ++i)
        an = (an * 1ll * a) % m;

    unordered_map<int, int> vals;
    for (int q = 0, cur = b; q <= n; ++q) {
        vals[cur] = q;
        cur = (cur * 1ll * a) % m;
    }

    for (int p = 1, cur = 1; p <= n; ++p) {
        cur = (cur * 1ll * an) % m;
        if (vals.count(cur)) {
            int ans = n * p - vals[cur];
            return ans;
        }
    }
    return -1;
}


int main() {

    const int arraySize = 4;
    int a[arraySize] = { 1,  2,  3,  4 };


    cout << solve() << endl;
}


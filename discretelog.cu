#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include<stdio.h>
#include <cmath>
#include<stdlib.h>
#include<iostream>
#include <iomanip>

#include <stdio.h>

using namespace std;
typedef unsigned int uint;



static const int wholeArraySize = 4;
static const int blockSize = 1024;
static const int gridSize = 2;

__device__ bool lastBlock(int* counter) {
    __threadfence(); //ensure that partial result is visible by all blocks
    int last = 0;
    if (threadIdx.x == 0)
        last = atomicAdd(counter, 1);
    return __syncthreads_or(last == gridDim.x - 1);
}

__global__ void sumCommMultiBlock(
    int start, 
    int end, 
    int target, 
    int* gOut, 
    int* lastBlockCounter){

    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;

    int sum = 0;
    for (int i = gthIdx; i < end; i += gridSize)

        if (i = target) 
            sum += target;



    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
    if (lastBlock(lastBlockCounter)) {
        shArr[thIdx] = thIdx < gridSize ? gOut[thIdx] : 0;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size)
                shArr[thIdx] += shArr[thIdx + size];
            __syncthreads();
        }
        if (thIdx == 0)
            gOut[0] = shArr[0];
    }
}

int sumArray(int* arr) {
    int* dev_arr;

    //cout << "start alloc" << endl;
    hipMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    hipMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), hipMemcpyHostToDevice);

    int target;
    //hipMalloc((void**)&target, sizeof(int));
    //hipMemcpy(target, 10, sizeof(int), hipMemcpyHostToDevice);

    int out;
    int* dev_out;
    hipMalloc((void**)&dev_out, sizeof(int) * gridSize);

    int* dev_lastBlockCounter;
    hipMalloc((void**)&dev_lastBlockCounter, sizeof(int));
    hipMemset(dev_lastBlockCounter, 0, sizeof(int));

    sumCommMultiBlock << <gridSize, blockSize >> > (0, 2048, 10, dev_out, dev_lastBlockCounter);
    hipDeviceSynchronize();

    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_arr);
    hipFree(dev_out);

    return out;
}


int main() {

    const int arraySize = 4;
    int a[arraySize] = { 1,  2,  3,  4 };


    cout << sumArray(a) << endl;
}

